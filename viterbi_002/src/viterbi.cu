#include "hip/hip_runtime.h"
#include <iostream>
#include <iterator>
#include <stdio.h>
#include <vector>
#include <algorithm>
#include <ctime>

#include "viterbi.h"

#define NUM_STATES 64
#define NUM_INPUT_SYMBOLS 2
#define NUM_OUTPUT_SYMBOLS 4
#define NUM_BLOCKS (189)
#define TRACEBACK (128)
#define NUM_H 36288
#define NUM_D_BLOCKS 378
#define NUM_D_THREADS 96
#define NUM_D_BITS 756
#define NUM_L (7*5)
#define BLOCK_LEN (TRACEBACK + 2*NUM_L)

__constant__ const char outputs[NUM_STATES][NUM_INPUT_SYMBOLS] = { { 0, 3 }, {
		3, 0 }, { 1, 2 }, { 2, 1 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 }, { 3,
		0 }, { 0, 3 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, { 2, 1 },
		{ 1, 2 }, { 3, 0 }, { 0, 3 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, {
				2, 1 }, { 1, 2 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 },
		{ 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, {
				0, 3 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, { 1, 2 },
		{ 2, 1 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 }, { 0, 3 }, { 3, 0 }, {
				1, 2 }, { 2, 1 }, { 0, 3 }, { 3, 0 }, { 1, 2 }, { 2, 1 },
		{ 0, 3 }, { 3, 0 }, { 2, 1 }, { 1, 2 }, { 3, 0 }, { 0, 3 }, { 2, 1 }, {
				1, 2 }, { 3, 0 }, { 0, 3 }, };

typedef struct {
	unsigned int w;
	unsigned char prev;
} pm_t;

__global__ void calc_bm(char *encData,
		pm_t (*pm)[NUM_BLOCKS][BLOCK_LEN][NUM_STATES]) {
	int state = threadIdx.x;
	int block = blockIdx.x;

	int prevState = (state & (~32U)) << 1;
	int prevSymbol = (state >> 5) & 1;

	int offset = block * (BLOCK_LEN);
	unsigned int pm0 = 0;
	unsigned int pm1 = 0;

	for (int tms = 0; tms < (BLOCK_LEN); tms++) {
		int _tms = tms + offset;

		unsigned char data = encData[_tms];
		unsigned char en0 = (~data >> 2) & 1;
		unsigned char en1 = (~data >> 3) & 1;
		unsigned char d0 = (data ^ outputs[prevState][prevSymbol]);
		unsigned char d1 = (data ^ outputs[prevState + 1][prevSymbol]);

		unsigned int c0 = ((d0 >> 1) & en1) + (d0 & en0);
		unsigned int c1 = ((d1 >> 1) & en1) + (d1 & en0);

		unsigned int w0 = c0 + pm0;
		unsigned int w1 = c1 + pm1;

		(*pm)[block][tms][state] = {
			.w = w0 < w1 ? w0 : w1,
			.prev = w0 < w1 ? (unsigned char)(prevState) : (unsigned char)(prevState + 1)
		};

		__syncthreads();

		pm0 = (*pm)[block][tms][prevState].w;
		pm1 = (*pm)[block][tms][prevState + 1].w;
	}

}

__global__ void merge(pm_t (*pm)[NUM_BLOCKS][TRACEBACK][NUM_STATES]) {
	int state = threadIdx.x;

	for (int block = 1; block < NUM_BLOCKS; block++) {
		unsigned int w0 = 0;
		unsigned int w1 = 0;

		int prevState = (state & (~32U)) << 1;
		if (block > 0) {
			w0 = (*pm)[block - 1][TRACEBACK - 1][prevState].w;
			w1 = (*pm)[block - 1][TRACEBACK - 1][prevState + 1].w;
			(*pm)[block][TRACEBACK - 1][state].w += w0 < w1 ? w0 : w1;
		}

		__syncthreads();
	}

}

__global__ void traceback(char *decData,
		pm_t (*pm)[NUM_BLOCKS][BLOCK_LEN][NUM_STATES]) {

	int block = blockIdx.x;
	int offset = block * (TRACEBACK);

	unsigned int minIdx = 0;
//	unsigned int min = 0xffffffff;
//	int tms = BLOCK_LEN - 1;
//	for (int i = 0; i < NUM_STATES; i++) {
//		unsigned int w = (*pm)[block][tms][i].w;
//		if (min > w) {
//			min = w;
//			minIdx = i;
//		}
//	}

	for (int _tms = BLOCK_LEN - 1; _tms >= BLOCK_LEN - NUM_L; _tms--) {
			minIdx = (*pm)[block][_tms][minIdx].prev;
	}

	for (int _tms = BLOCK_LEN - NUM_L - 1; _tms >= NUM_L; _tms--) {
		decData[_tms + offset - NUM_L] = (minIdx >> 5) & 1;
		minIdx = (*pm)[block][_tms][minIdx].prev;
	}
}

__global__ void deint_symbol(const char* bitset, char* tmp, int frame,
		int* deint_h) {
	int it = threadIdx.x;
	int block = blockIdx.x;
	int offset = block * NUM_D_THREADS + it;

	if (frame % 2 == 0) {
		tmp[offset] = bitset[deint_h[offset]];
	} else {
		tmp[deint_h[offset]] = bitset[offset];
	}
}

__global__ void deint_bit(const char* tmp, char* result, int* bit_table) {
	int inner = threadIdx.x;
	int outer = blockIdx.x;
	result[bit_table[inner] - 1 + outer * 756] = tmp[inner + outer * 756];
}

static pm_t (*pm)[NUM_BLOCKS][BLOCK_LEN][NUM_STATES];
static char *encData;
static char *decData;
static char *deint_tmp;
static char *deint_data;
static char *deint_output;
static int *deint_h;
static int *deint_bit_table;

hipStream_t viterbi_stream;
hipStream_t deint_stream;

void gpu_deint_init() {
	hipStreamCreate(&deint_stream);
	hipMalloc(&deint_tmp, NUM_D_BLOCKS * NUM_D_THREADS);
	hipMalloc(&deint_data, NUM_D_BLOCKS * NUM_D_THREADS);
	hipMalloc(&deint_output, NUM_D_BLOCKS * NUM_D_THREADS);
	hipMalloc(&deint_h, NUM_D_BLOCKS * NUM_D_THREADS * sizeof(int));
	hipMalloc(&deint_bit_table, NUM_D_BITS * sizeof(int));

	hipMemcpy(deint_h, H, NUM_D_THREADS * NUM_D_BLOCKS * sizeof(int),
			hipMemcpyHostToDevice);
	hipMemcpy(deint_bit_table, bit_table, NUM_D_BITS * sizeof(int),
			hipMemcpyHostToDevice);
}

void gpu_deint_free() {
	hipStreamDestroy(deint_stream);
	hipFree(deint_tmp);
	hipFree(deint_data);
	hipFree(deint_output);
	hipFree(deint_h);
	hipFree(deint_bit_table);
}

void gpu_viterbi_init() {
	hipStreamCreate(&viterbi_stream);
	hipMalloc(&pm,
	NUM_BLOCKS * (BLOCK_LEN) * NUM_STATES * sizeof(pm_t));
	hipMalloc(&encData, (BLOCK_LEN) * NUM_BLOCKS);
	hipMalloc(&decData, TRACEBACK * NUM_BLOCKS);
}

void gpu_viterbi_free() {
	hipFree(pm);
	hipFree(encData);
	hipFree(decData);
	hipStreamDestroy(viterbi_stream);
}

void gpu_viterbi_decode(const char* data, char* output) {

	static char encData_tmp[(BLOCK_LEN) * NUM_BLOCKS] = {0,};
	char decData_tmp[TRACEBACK * NUM_BLOCKS];

	for (int b = 0; b < NUM_BLOCKS; b++) {
		for (int i = 0; i < TRACEBACK + NUM_L; i++) {

			int d = (data[(b * TRACEBACK + i) * 2] & 1) << 1
					| (data[(b * TRACEBACK + i) * 2 + 1] & 1);
			d |= (data[(b * TRACEBACK + i) * 2] & 2) << 2
					| (data[(b * TRACEBACK + i) * 2 + 1] & 2) << 1;

			encData_tmp[b * (BLOCK_LEN) + i + NUM_L] = d;
		}
		if (b > 0) {
//			memcpy(&encData_tmp[(b-1)*BLOCK_LEN + TRACEBACK + NUM_L], &encData_tmp[b*BLOCK_LEN + NUM_L], NUM_L);
			memcpy(&encData_tmp[(b) * (BLOCK_LEN)], &encData_tmp[(b - 1)
						* (BLOCK_LEN) + TRACEBACK], NUM_L);
		}
	}

	hipMemcpy(encData, encData_tmp, (BLOCK_LEN) * NUM_BLOCKS,
			hipMemcpyHostToDevice);
	calc_bm<<<dim3(NUM_BLOCKS), dim3(NUM_STATES), 0, viterbi_stream>>>(encData,
			pm);
	hipStreamSynchronize(viterbi_stream);
//	merge<<<dim3(1), dim3(NUM_STATES), 0, viterbi_stream>>>(pm);
//	hipStreamSynchronize(viterbi_stream);
	traceback<<<NUM_BLOCKS, 1, 0, viterbi_stream>>>(decData, pm);
	hipStreamSynchronize(viterbi_stream);

	hipMemcpy(decData_tmp, decData, TRACEBACK * NUM_BLOCKS,
			hipMemcpyDeviceToHost);

	memcpy(encData_tmp, &encData_tmp[(NUM_BLOCKS-1)*BLOCK_LEN + TRACEBACK], NUM_L);

	int count = 0;
	for (int i = 0; i < TRACEBACK * NUM_BLOCKS; i += 8) {
		output[i >> 3] = 0;
		for (int j = 7; j >= 0; j--) {
			output[i >> 3] |= (decData_tmp[count++] & 1) << j;
		}
	}

}

void gpu_deinterleave(const char* data, char* output, int frame) {

	hipMemcpy(deint_data, data, NUM_D_THREADS * NUM_D_BLOCKS,
			hipMemcpyHostToDevice);

	deint_symbol<<<NUM_D_BLOCKS, NUM_D_THREADS, 0, deint_stream>>>(deint_data,
			deint_tmp, frame, deint_h);
	hipStreamSynchronize(deint_stream);
	deint_bit<<<48, 756, 0, deint_stream>>>(deint_tmp, deint_output,
			deint_bit_table);
	hipStreamSynchronize(deint_stream);

	hipMemcpy(output, deint_output, NUM_D_THREADS * NUM_D_BLOCKS,
			hipMemcpyDeviceToHost);
}
